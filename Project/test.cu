#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__device__ const int MAX_DEGREE = 4;

__device__ void sortEdges(int* edges, int* sorted)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x + y * gridDim.x * blockDim.x;
	int n1 = edges[i * 2];
	int n2 = edges[i * 2 + 1];
	
	int* arrStart = &sorted[n1 * MAX_DEGREE];
	int retnVal = 1;
	while(retnVal != 0)
		retnVal = atomicCAS(arrStart, 0, n2);
}


//HACK: This will be incredibly slow  on CUDA!
__device__ int findNext(int* edges, int numEdge, int v, int* destination)
{
	int count = 0;

	for(int i = 0; i < numEdge * 2; i+=2)
	{
		if(edges[i] == v)
			destination[count++] = edges[i + 1];
	}

	return count;
}

__device__ void pushQueue(int element, int* queue, int queueSize, int* head, int* tail)
{
	queue[*tail] = element;
	*tail = (*tail + 1) % queueSize;
}

__device__ int popQueue(int* queue, int queueSize, int* head, int* tail)
{
	int retn = queue[*head];
	*head = (*head + 1) % queueSize;
	
	return retn;
}

__device__ void pushStack(int element, int* stack, int* head)
{
	stack[*head] = element;
	*head = *head + 1;
}

__device__ int popStack(int* stack, int* head)
{
	*head = *head - 1;
	int retn = stack[*head];
	
	return retn;
}

const int ELEMENTS = 64;
const int S_SIZE = ELEMENTS;
const int P_SIZE = ELEMENTS;
const int P_LIST_SIZE = ELEMENTS;
const int PATH_SIZE = ELEMENTS;
const int D_SIZE = ELEMENTS;
const int Q_SIZE = ELEMENTS;


__device__ void doAlg(int numVert, int* edges, int numEdges, int* BC, int* glob)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.x + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	int PTR_OFFSET = idx * (S_SIZE + (P_SIZE * MAX_DEGREE) + D_SIZE + Q_SIZE + PATH_SIZE);
	
	int* S = &glob[PTR_OFFSET];
	int S_head = 0;
	PTR_OFFSET += S_SIZE;
	
	int* P = &glob[PTR_OFFSET];
	//Blank the previous items
	for(int i = 0; i < P_SIZE; i++)
		for(int j = 0; j < MAX_DEGREE; j++)
		{
			P[i * P_SIZE + j] = -1;
		}
	PTR_OFFSET += P_SIZE * MAX_DEGREE;

	int* pathCount = &glob[PTR_OFFSET];
	pathCount[idx] = 1;
	PTR_OFFSET += PATH_SIZE;

	int* d = &glob[PTR_OFFSET];
	for(int i = 0; i < D_SIZE; i++)
	{
		d[i] = -1;
	}
	d[idx] = 0;
	PTR_OFFSET += D_SIZE;
	
	int* Q = &glob[PTR_OFFSET];
	int Q_head = 0;
	int Q_tail = 0;
	PTR_OFFSET += Q_SIZE;
	
	pushQueue(idx, Q, Q_SIZE, &Q_head, &Q_tail);

	while(Q_head != Q_tail)
	{
		int v = popQueue(Q, Q_SIZE, &Q_head, &Q_tail);
		pushStack(v, S, &S_head);

		int w[8];
		int edgeCount = findNext(edges, numEdges, v, w);

		for(int i = 0; i < edgeCount; i++)
		{
			int wNode = w[i];
			if(d[wNode] < 0)
			{
				pushQueue(wNode, Q, Q_SIZE, &Q_head, &Q_tail);
				d[wNode] = d[v] + 1;
			}
			
			if(d[wNode] == d[v] + 1)
			{
				pathCount[wNode] = pathCount[wNode] + pathCount[v];
				
				//Append v to the PrevNode list
				for(int j = 0; j < MAX_DEGREE; j++)
				{
					if(P[wNode * P_SIZE + j] < 0)
					{
						P[wNode * P_SIZE + j] = v;
						break;
					}
				}
			}
		}

	}
	
	float* dep = (float*)&glob[PTR_OFFSET];
	
	while(S_head > 0)
	{
		int w = popStack(S, &S_head);
		
		//Loop through each v in P[w]
		for(int i = 0; i < MAX_DEGREE; i++)
		{
			int v = P[0];//P[w * P_SIZE + i];
			//dep[v] = dep[v] + (pathCount[v]/pathCount[w]) * (1 + dep[w]);
		}
		
		if(w != idx)
		{
			//TODO: Atomic
			BC[w] = 1;//BC[w] + dep[w];
		}
	}
	
}

__global__ void betweennessCentrality(int numVert, int numEdges, int *edges, int* BC, int* glob)
{
	extern __shared__ int path[];
	
	//sortEdges(edges, path);
	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.x + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	doAlg(numVert, edges, numEdges, BC, glob);

		
}

int main()
{
	const int elements = 64;

	//cudaProfilerStart();
	int *d_mem;
	int *h_edge;
	int *d_edge;
	int *d_bc;
	int *h_bc;
	int *d_glob;
	
	hipMalloc((void**)&d_mem, sizeof(int) * elements);
	
	h_edge = (int*)malloc(sizeof(int) * elements * 2);
	hipMalloc((void**)&d_edge, sizeof(int) * elements * 2);

	h_bc = (int*)malloc(sizeof(int) * elements);
	hipMalloc((void**)&d_bc, sizeof(int) * elements);

	hipMalloc((void**)&d_glob, sizeof(int) * elements * elements * 20);
	
	//Init edges
	for(int i = 0; i < elements; i++)
	{
		h_edge[i * 2] = i % elements;
		h_edge[i * 2 + 1] = (i + 1) % elements;
	}
	hipMemcpy(d_edge, h_edge, sizeof(int) * elements * 2, hipMemcpyHostToDevice);
	
	dim3 block(8,8);
	dim3 grid(elements / 64);
	//test<<<grid,block>>>(d_mem);
	betweennessCentrality<<<grid,block>>>(elements, elements, d_edge, d_bc, d_glob);
	hipError_t error = hipGetLastError();
	
	int* h_mem = (int*)malloc(sizeof(int) * elements);
	hipMemcpy(h_mem, d_mem, sizeof(int) * elements, hipMemcpyDeviceToHost);
	hipMemcpy(h_bc, d_bc, sizeof(int) * elements, hipMemcpyDeviceToHost);
	

	for(int i = 0; i < elements; i++)
	{
		cout << h_bc[i] << endl;
	}
	//cout<<elements<<endl;
	
	//cudaProfilerStop();
	
	hipDeviceReset();
	cout << hipGetErrorString(error) << endl;
	
	return 0;
}
