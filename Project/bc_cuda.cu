#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

const int BLOCK_WIDTH = 2;
const int BLOCK_HEIGHT = 2;
const int DEFAULT_ELE = 16;
extern __shared__ int shmem[];

typedef struct __align__(8) linkNode
{
	int edge;
	int next;
} linkNode;

//Create CSR edge storage
__global__ void convertEdges(int* edges, int numEdge, int numVert, int* newArrays)
{
	int* val = newArrays;
	int* col = &val[numEdge];// + numEdge;
	int* row = &col[numEdge];

	int lastRow = -1;
	int rowNum = 0;
	for(int i = 0; i < numEdge; i++)
	{
		int v1 = edges[i * 2];//Row
		int v2 = edges[i * 2 + 1];//Col

		val[i] = 1;//Weight
		col[i] = v2;

		if(lastRow != v1)
		{
			//Make sure to skip unconnected vertices
			int diff = v1 - lastRow;
			for(int j = 0; j < diff - 1; j++)
				row[rowNum++] = i;
			row[rowNum++] = i;
			lastRow = v1;
		}
	}

	//Fill in final row columns
	for(int i = rowNum; i <= numVert; i++)
		row[i] = numEdge;
}

__device__ int findNext(int* edges, int numEdge, int numVert, int v, int* destination)
{
	int* val = edges;
	int* col = &val[numEdge];
	int* row = &col[numEdge];

	int idx = row[v];
	int nextIdx = row[v + 1];

	int count = 0;
	for(int i = idx; i < nextIdx; i++)
	{
		destination[count++] = col[i];
	}

	return count;
}

__device__ void pushQueue(int element, int* queue, int queueSize, unsigned int* head, unsigned int* tail)
{
	int idx = atomicInc(tail, queueSize);
	queue[idx] = element;
}

__device__ int popQueue(int* queue, int queueSize, unsigned int* head, unsigned int* tail)
{
	int retn;
	if(*head == *tail) retn = -1;
	else
	{
		int idx = atomicInc(head, queueSize);
		retn = queue[idx];
	}
	
	return retn;
}

__device__ void pushStack(int element, int* stack, int* head)
{
	int idx = atomicAdd(head, 1);
	stack[idx] = element;
}

__device__ int popStack(int* stack, int* head)
{
	int retn;
	if(*head == 0) retn = -1;
	else
	{
		int idx = atomicSub(head, 1);
		retn = stack[idx - 1];
	}
	
	return retn;
}

__device__ int ELEMENTS = DEFAULT_ELE;
__device__ int S_SIZE;
__device__ int P_SIZE;
__device__ int PATH_SIZE;
__device__ int D_SIZE;
__device__ int Q_SIZE;

__device__ void doAlg(int numVert, int* edges, int numEdges, linkNode* pList, float* BC, int* glob, float* globDep)
{
	S_SIZE = numVert;
	P_SIZE = numEdges + numVert;
	D_SIZE = numVert;
	Q_SIZE = numVert;
	PATH_SIZE = numVert;
	ELEMENTS = numVert;

	int block_idx = gridDim.x * blockIdx.y + blockIdx.x;
	int localIdx = threadIdx.x + threadIdx.y * blockDim.x;

	unsigned int PTR_OFFSET = block_idx * (S_SIZE + D_SIZE + Q_SIZE + PATH_SIZE);
	
	int* S = &glob[PTR_OFFSET];
	int* S_head = &shmem[4];
	*S_head = 0;
	PTR_OFFSET += S_SIZE;
	
	linkNode* P = &pList[block_idx * P_SIZE];
	//Blank the previous items
	for(int i = 0; i < P_SIZE && localIdx == 0; i++)
	{
		P[i].edge = -1;
		P[i].next = -1;
	}

	int* pathCount = &glob[PTR_OFFSET];
	for(int i = 0; i < PATH_SIZE; i++)
	{
		pathCount[i] = 0;
	}
	pathCount[block_idx] = 1;
	PTR_OFFSET += PATH_SIZE;

	int* d = &glob[PTR_OFFSET];
	for(int i = 0; i < D_SIZE; i++)
	{
		d[i] = -1;
	}
	d[block_idx] = 0;
	PTR_OFFSET += D_SIZE;
	
	int* Q = &glob[PTR_OFFSET];
	unsigned int* Q_head = (unsigned int*)&shmem[2];
	unsigned int* Q_tail = (unsigned int*)&shmem[3];
	if(localIdx == 0)
	{
		*Q_head = 0;
		*Q_tail = 0;
	}
	PTR_OFFSET += Q_SIZE;
	
	if(localIdx == 0)
		pushQueue(block_idx, Q, Q_SIZE, Q_head, Q_tail);

	int* front = &shmem[0];
	int* nextFront = &shmem[1];
	if(localIdx == 0)
	{
		*front = 1;
		*nextFront = 0;
	}

	int* testCnt = &shmem[5];
	*testCnt = 0;

	while(*Q_head != *Q_tail || *front != 0 || *nextFront != 0)
	{
		__syncthreads();
		if(*front == 0 && localIdx == 0)
		{
			*front = *nextFront;
			*nextFront = 0;
		}
		__syncthreads();
		int v = -1;
		int atmSub = atomicSub(front, 1);
		if(atmSub > 0)
			v = popQueue(Q, Q_SIZE, Q_head, Q_tail);
		else
			atomicAdd(front, 1);//Don't let the counter go below 0
		if(v < 0) continue;

		pushStack(v, S, S_head);

		int w[1024];
		int edgeCount = findNext(edges, numEdges, numVert, v, w);

		for(int i = 0; i < edgeCount; i++)
		{
			int wNode = w[i];
			if(d[wNode] < 0 && atomicCAS(&d[wNode], -1, d[v] + 1) == -1)
			{
				pushQueue(wNode, Q, Q_SIZE, Q_head, Q_tail);
				atomicAdd(nextFront, 1);//Frontier expands
			}

			if(d[wNode] == d[v] + 1)
			{
				atomicAdd(&pathCount[wNode], pathCount[v]);
				//pathCount[wNode] = pathCount[wNode] + pathCount[v];
				
				//Append v to the PrevNode list

				//Find the next empty slot. Start after the initial lookup indices
				if(atomicCAS(&P[wNode].edge, -1, v) < 0);
				else
				{
					atomicAdd(testCnt, 1);
					int empty;
					int casToken = 0;
					for(empty = numVert; casToken != -1; empty++)
					{
						casToken = atomicCAS(&P[empty].edge, -1, v);
					}
					empty--;

					linkNode* j = &P[wNode];
					//while(j !=  NULL)
					while(atomicCAS(&(j->next), -1, empty) >= 0)
					{
						j = &P[j->next];
					}
				}
				
				
			}
		}

	}
	
	float* dep = &globDep[block_idx * numVert];

	__syncthreads();
	
	while(*S_head > 0 && localIdx == 0)
	{
		int w = popStack(S, S_head);
		if(w == block_idx) continue;
		
		//Loop through each v in P[w]
		linkNode* node = &P[w];
		while(node != NULL)
		{
			int v = node->edge;
			if(v < 0) continue;

			dep[v] = dep[v] + ((float)pathCount[v]/(float)pathCount[w]) * (1 + dep[w]);

			node = node->next < 0 ? NULL : &P[node->next];
		}
		
		if(w != block_idx)
		{
			atomicAdd(&BC[w], dep[w]);
		}
	}
	
}

__global__ void betweennessCentrality(int numVert, int numEdges, int *edges, linkNode* pList, float* BC, int* glob, float* dep)
{
	//sortEdges(edges, path);
	int block_idx = blockIdx.x + blockIdx.y * gridDim.x;

	if(block_idx >= numVert) return;

	shmem[0] = 0;

	__syncthreads();

	doAlg(numVert, edges, numEdges, pList, BC, glob, dep);

		
}

int edgeCompare(const void* a, const void* b)
{
	int* av1 = (int*)a;
	int* av2 = av1 + 1;
	int* bv1 = (int*)b;
	int* bv2 = bv1 + 1;

	if(*av1 < *bv1) return -1;
	else if(*av1 > *bv1) return 1;
	else if(*av2 < *bv2) return -1;
	else if(*av2 > *bv2) return 1;
	else return 0;
}

int main(int argc, char* argv[])
{
	int elements = DEFAULT_ELE;

	//cudaProfilerStart();
	int *d_mem;
	int *h_edge;
	int *d_edge;
	int *d_optEdge;
	linkNode* pList;
	float *d_bc;
	float *h_bc;
	int *d_glob;
	float *d_dep;

	int numVert = elements;
	int numEdge = elements - 1;

	if(argc < 2)
	{

		FILE *grFile;
		grFile = fopen("test.gr", "w");
		fprintf(grFile, "p %d %d d u 0\n", numVert, numEdge);
		h_edge = (int*)malloc(sizeof(int) * numEdge * 2);
		
		//Init edges
		for(int i = 0; i < numEdge; i++)
		{
			h_edge[i * 2] = i % numVert;
			h_edge[i * 2 + 1] = (i + 1) % numVert;
			fprintf(grFile, "%d %d\n", h_edge[i * 2], h_edge[i * 2 + 1]);
		}
		fclose(grFile);
	}
	else
	{
		FILE *grFile;
		char buff[1024];
		int edgeCnt = 0;
		grFile = fopen(argv[1], "r");
		while(!feof(grFile))
		{
			fgets(buff, 1024, grFile);
			if(feof(grFile))break;

			//This is the  "problem" line
			if(buff[0] == 'p')
			{
				char* token = strtok(buff, " ");

				token = strtok(NULL, " ");
				token = strtok(NULL, " ");
				numVert = atoi(token);
				
				token = strtok(NULL, " ");
				numEdge = atoi(token) * 2;

				h_edge = (int*)malloc(sizeof(int) * numEdge * 2);

			}
			else if(buff[0] == '#' || buff[0] == 'c')
				continue;
			else if(buff[0] == 'a')
			{
				char* token = strtok(buff, " ");

				//Skip 'a'
				token = strtok(NULL, " ");
				int e1 = atoi(token) - 1;
				h_edge[edgeCnt] = e1;
				h_edge[edgeCnt + 3] = e1;
				token = strtok(NULL, " ");
				int e2 = atoi(token) - 1;
				h_edge[edgeCnt + 1] = e2;
				h_edge[edgeCnt + 2] = e2;
				edgeCnt += 4;
			}
		}
	}

	//Sort the arrays for CSR
	qsort(h_edge, numEdge, sizeof(int) * 2, edgeCompare);
	
	long totalMem = 0;
	hipMalloc((void**)&d_optEdge, sizeof(int) * (numVert + 1) * (numEdge * 2));
	totalMem += sizeof(int) * (numVert + 1) * (numEdge * 2);


	struct timeval totalStart, totalEnd;
	gettimeofday(&totalStart, NULL);

	//First convert edges
	hipMalloc((void**)&d_edge, sizeof(int) * numEdge * 2);
	//totalMem += sizeof(int) * numEdge * 2;
	hipMemcpy(d_edge, h_edge, sizeof(int) * numEdge * 2, hipMemcpyHostToDevice);
	convertEdges<<<1,1>>>(d_edge, numEdge, numVert, d_optEdge);
	hipDeviceSynchronize();
	hipFree(d_edge);

	//Test code
	int* test = (int*)malloc(sizeof(int) * (numVert + 1) * (numEdge * 2));
	hipMemcpy(test, d_optEdge, sizeof(int) * (numVert + 1) * (numEdge * 2), hipMemcpyDeviceToHost);
	int* val = test;
	int* col = test + numEdge;
	int* row = col + numEdge;
	

	h_bc = (float*)malloc(sizeof(float) * numVert);
	hipMalloc((void**)&d_mem, sizeof(int) * numVert);
	totalMem += sizeof(int) * numVert;
	
	hipMalloc((void**)&d_bc, sizeof(float) * numVert);
	hipMemset(d_bc, 0, sizeof(float) * numVert);
	totalMem += sizeof(float) * numVert;

	hipMalloc((void**)&d_glob, sizeof(int) * numVert * (numVert * 5));
	totalMem += sizeof(int) * numVert * ((numVert * 5));

	hipMalloc((void**)&pList, sizeof(linkNode) * numVert * (numVert + numEdge));
	totalMem += sizeof(linkNode) * numVert * (numVert + numEdge);

	hipMalloc((void**)&d_dep, sizeof(float) * numVert * numVert);
	totalMem += sizeof(float) * numVert * numVert;
	
	dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	int gridSize = numVert;//ceil(numVert / (float)(BLOCK_WIDTH * BLOCK_HEIGHT));
	dim3 grid(gridSize);

	struct timeval start, end;

	gettimeofday(&start, NULL);
	betweennessCentrality<<<grid,block, 512>>>(numVert, numEdge, d_optEdge, pList, d_bc, d_glob, d_dep);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	long elapsed = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);

	hipError_t error = hipGetLastError();
	
	int* h_mem = (int*)malloc(sizeof(int) * numVert);
	hipMemcpy(h_mem, d_mem, sizeof(int) * numVert, hipMemcpyDeviceToHost);
	hipMemcpy(h_bc, d_bc, sizeof(float) * numVert, hipMemcpyDeviceToHost);

	gettimeofday(&totalEnd, NULL);
	long totalElapsed = (totalEnd.tv_sec * 1000000 + totalEnd.tv_usec) - 
		( totalStart.tv_sec * 1000000 + totalStart.tv_usec); 

	for(int i = 0; i < numVert; i++)
	{
		cout << h_bc[i] << endl;
	}
	//cout<<elements<<endl;
	
	//cudaProfilerStop();
	
	hipDeviceReset();
	cout << hipGetErrorString(error) << endl;
	cout << "Mem Used: " << totalMem << endl;
	cout << "Time(usec): " << elapsed << endl;
	cout << "Total Time(usec): " << totalElapsed << endl;
	
	return 0;
}
