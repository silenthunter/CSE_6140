#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

const int BLOCK_WIDTH = 2;
const int BLOCK_HEIGHT = 2;
const int DEFAULT_ELE = 16;
extern __shared__ int shmem[];
const int LOCK = 0;
const int WARP_SIZE = 32;

typedef struct __align__(8) linkNode
{
	int edge;
	linkNode* next;
} linkNode;

/*__device__ void lock()
{
	int localId = threadIdx.x + threadIdx.y * blockDim.x;
	for(int i = 0; i < blockDim.x * blockDim.y; i++)
		if(localId == i)
			while(atomicCAS(&shmem[LOCK], 0, 1) == 1);
}

__device__ void unlock()
{
	atomicExch(&shmem[LOCK], 0);
}*/

//HACK: This will be incredibly slow  on CUDA!
__device__ int findNext(int* edges, int numEdge, int v, int* destination)
{
	int count = 0;

	for(int i = 0; i < numEdge * 2; i+=2)
	{
		if(edges[i] == v)
			destination[count++] = edges[i + 1];
	}

	return count;
}

__device__ void pushQueue(int element, int* queue, int queueSize, unsigned int* head, unsigned int* tail)
{
	int idx = atomicInc(tail, queueSize);
	queue[idx] = element;
}

__device__ int popQueue(int* queue, int queueSize, unsigned int* head, unsigned int* tail)
{
	int retn;
	if(*head == *tail) retn = -1;
	else
	{
		int idx = atomicInc(head, queueSize);
		retn = queue[idx];
	}
	
	return retn;
}

__device__ void pushStack(int element, int* stack, int* head)
{
	int idx = atomicAdd(head, 1);
	stack[idx] = element;
}

__device__ int popStack(int* stack, int* head)
{
	int retn;
	if(*head == 0) retn = -1;
	else
	{
		int idx = atomicSub(head, 1);
		retn = stack[idx - 1];
	}
	
	return retn;
}

__device__ int ELEMENTS = DEFAULT_ELE;
__device__ int S_SIZE;
__device__ int P_SIZE;
__device__ int PATH_SIZE;
__device__ int D_SIZE;
__device__ int Q_SIZE;

__device__ void doAlg(int numVert, int* edges, int numEdges, linkNode* pList, float* BC, int* glob, float* globDep)
{
	S_SIZE = numVert;
	P_SIZE = numEdges + numVert;
	D_SIZE = numVert;
	Q_SIZE = numVert;
	PATH_SIZE = numVert;
	ELEMENTS = numVert;

	int block_idx = gridDim.x * blockIdx.y + blockIdx.x;
	int localIdx = threadIdx.x + threadIdx.y * blockDim.x;

	unsigned int PTR_OFFSET = block_idx * (S_SIZE + D_SIZE + Q_SIZE + PATH_SIZE);
	
	int* S = &glob[PTR_OFFSET];
	int* S_head = &shmem[4];
	*S_head = 0;
	PTR_OFFSET += S_SIZE;
	
	linkNode* P = &pList[block_idx * P_SIZE];
	//Blank the previous items
	for(int i = 0; i < P_SIZE; i++)
	{
		P[i].edge = -1;
		P[i].next = NULL;
	}

	int* pathCount = &glob[PTR_OFFSET];
	for(int i = 0; i < PATH_SIZE; i++)
	{
		pathCount[i] = 0;
	}
	pathCount[block_idx] = 1;
	PTR_OFFSET += PATH_SIZE;

	int* d = &glob[PTR_OFFSET];
	for(int i = 0; i < D_SIZE; i++)
	{
		d[i] = -1;
	}
	d[block_idx] = 0;
	PTR_OFFSET += D_SIZE;
	
	int* Q = &glob[PTR_OFFSET];
	unsigned int* Q_head = (unsigned int*)&shmem[2];
	unsigned int* Q_tail = (unsigned int*)&shmem[3];
	*Q_head = 0;
	*Q_tail = 0;
	PTR_OFFSET += Q_SIZE;
	
	if(localIdx == 0)
		pushQueue(block_idx, Q, Q_SIZE, Q_head, Q_tail);

	unsigned int* front = (unsigned int*)&shmem[0];
	unsigned int* nextFront = (unsigned int*)&shmem[1];
	*front = 1;
	*nextFront = 0;

	while(*Q_head != *Q_tail || *front != 0 || *nextFront != 0)
	{
		__syncthreads();
		if(*front == 0 && localIdx == 0)
		{
			*front = *nextFront;
			*nextFront = 0;
		}
		__syncthreads();
		int v = -1;
		if(atomicDec(front, Q_SIZE))
			v = popQueue(Q, Q_SIZE, Q_head, Q_tail);
		//if(v < 0) continue;

		pushStack(v, S, S_head);

		int w[1024];
		int edgeCount = findNext(edges, numEdges, v, w);

		for(int i = 0; i < edgeCount; i++)
		{
			int wNode = w[i];
			if(d[wNode] < 0)
			{

				pushQueue(wNode, Q, Q_SIZE, Q_head, Q_tail);
				atomicAdd(nextFront, 1);//Frontier expands
				d[wNode] = d[v] + 1;
			}

			if(d[wNode] == d[v] + 1)
			{
				pathCount[wNode] = pathCount[wNode] + pathCount[v];
				
				//Append v to the PrevNode list

				//Find the next empty slot. Start after the initial lookup indices
				if(P[wNode].edge < 0) P[wNode].edge = v;
				else
				{
					linkNode* empty;
					for(empty = &P[numVert]; empty->edge >= 0; empty++);

					linkNode* j = &P[wNode];
					linkNode* last = NULL;
					while(j !=  NULL)
					{
						last = j;
						j = j->next;
					}
					last->next = empty;
					empty->edge = v;
				}
				
				
			}
		}

	}
	
	float* dep = &globDep[block_idx * numVert];

	__syncthreads();
	
	while(*S_head > 0 && localIdx == 0)
	{
		int w = popStack(S, S_head);
		
		//Loop through each v in P[w]
		linkNode* node = &P[w];
		while(node != NULL)
		{
			int v = node->edge;
			node = node->next;
			if(v < 0) continue;

			dep[v] = dep[v] + ((float)pathCount[v]/(float)pathCount[w]) * (1 + dep[w]);
		}
		
		if(w != block_idx)
		{
			atomicAdd(&BC[w], dep[w]);
		}
	}
	
}

__global__ void betweennessCentrality(int numVert, int numEdges, int *edges, linkNode* pList, float* BC, int* glob, float* dep)
{
	//sortEdges(edges, path);
	int block_idx = blockIdx.x + blockIdx.y * gridDim.x;

	if(block_idx >= numVert) return;

	BC[block_idx] = 0.0f;
	shmem[0] = 0;

	__syncthreads();

	doAlg(numVert, edges, numEdges, pList, BC, glob, dep);

		
}

int main(int argc, char* argv[])
{
	int elements = DEFAULT_ELE;

	//cudaProfilerStart();
	int *d_mem;
	int *h_edge;
	int *d_edge;
	linkNode* pList;
	float *d_bc;
	float *h_bc;
	int *d_glob;
	float *d_dep;

	int numVert = elements;
	int numEdge = elements - 1;

	if(argc < 2)
	{

		FILE *grFile;
		grFile = fopen("test.gr", "w");
		fprintf(grFile, "p %d %d d u 0\n", numVert, numEdge);
		h_edge = (int*)malloc(sizeof(int) * numEdge * 2);
		
		//Init edges
		for(int i = 0; i < numEdge; i++)
		{
			h_edge[i * 2] = i % numVert;
			h_edge[i * 2 + 1] = (i + 1) % numVert;
			fprintf(grFile, "%d %d\n", h_edge[i * 2], h_edge[i * 2 + 1]);
		}
		fclose(grFile);
	}
	else
	{
		FILE *grFile;
		char buff[1024];
		int edgeCnt = 0;
		grFile = fopen(argv[1], "r");
		while(!feof(grFile))
		{
			fgets(buff, 1024, grFile);
			if(feof(grFile))break;

			//This is the  "problem" line
			if(buff[0] == 'p')
			{
				char* token = strtok(buff, " ");

				token = strtok(NULL, " ");
				token = strtok(NULL, " ");
				numVert = atoi(token);
				
				token = strtok(NULL, " ");
				numEdge = atoi(token) * 2;

				h_edge = (int*)malloc(sizeof(int) * numEdge * 2);

			}
			else if(buff[0] == '#' || buff[0] == 'c')
				continue;
			else if(buff[0] == 'a')
			{
				char* token = strtok(buff, " ");

				//Skip 'a'
				token = strtok(NULL, " ");
				int e1 = atoi(token) - 1;
				h_edge[edgeCnt] = e1;
				h_edge[edgeCnt + 3] = e1;
				token = strtok(NULL, " ");
				int e2 = atoi(token) - 1;
				h_edge[edgeCnt + 1] = e2;
				h_edge[edgeCnt + 2] = e2;
				edgeCnt += 4;
			}
		}
	}
	struct timeval totalStart, totalEnd;
	gettimeofday(&totalStart, NULL);

	long totalMem = 0;
	h_bc = (float*)malloc(sizeof(float) * numVert);
	hipMalloc((void**)&d_mem, sizeof(int) * numVert);
	totalMem += sizeof(int) * numVert;
	
	hipMalloc((void**)&d_edge, sizeof(int) * numEdge * 2);
	totalMem += sizeof(int) * numEdge * 2;

	hipMalloc((void**)&d_bc, sizeof(float) * numVert);
	totalMem += sizeof(float) * numVert;

	hipMalloc((void**)&d_glob, sizeof(int) * numVert * (numVert * 5));
	totalMem += sizeof(int) * numVert * ((numVert * 5));

	hipMalloc((void**)&pList, sizeof(linkNode) * numEdge * (numVert + numVert));
	totalMem += sizeof(linkNode) * numEdge * (numVert + numVert);

	hipMalloc((void**)&d_dep, sizeof(float) * numVert * numVert);
	totalMem += sizeof(float) * numVert * numVert;

	hipMemcpy(d_edge, h_edge, sizeof(int) * numEdge * 2, hipMemcpyHostToDevice);
	
	dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	int gridSize = numVert;//ceil(numVert / (float)(BLOCK_WIDTH * BLOCK_HEIGHT));
	dim3 grid(gridSize);

	struct timeval start, end;

	gettimeofday(&start, NULL);
	betweennessCentrality<<<grid,block, 20>>>(numVert, numEdge, d_edge, pList, d_bc, d_glob, d_dep);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	long elapsed = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);

	hipError_t error = hipGetLastError();
	
	int* h_mem = (int*)malloc(sizeof(int) * numVert);
	hipMemcpy(h_mem, d_mem, sizeof(int) * numVert, hipMemcpyDeviceToHost);
	hipMemcpy(h_bc, d_bc, sizeof(float) * numVert, hipMemcpyDeviceToHost);

	gettimeofday(&totalEnd, NULL);
	long totalElapsed = (totalEnd.tv_sec * 1000000 + totalEnd.tv_usec) - 
		( totalStart.tv_sec * 1000000 + totalStart.tv_usec); 

	for(int i = 0; i < numVert; i++)
	{
		cout << h_bc[i] << endl;
	}
	//cout<<elements<<endl;
	
	//cudaProfilerStop();
	
	hipDeviceReset();
	cout << hipGetErrorString(error) << endl;
	cout << "Mem Used: " << totalMem << endl;
	cout << "Time(usec): " << elapsed << endl;
	cout << "Total Time(usec): " << totalElapsed << endl;
	
	return 0;
}
