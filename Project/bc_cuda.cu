#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__device__ const int MAX_DEGREE = 50;

const int BLOCK_WIDTH = 2;
const int BLOCK_HEIGHT = 2;
const int DEFAULT_ELE = 1024;

__device__ void sortEdges(int* edges, int* sorted)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x + y * gridDim.x * blockDim.x;
	int n1 = edges[i * 2];
	int n2 = edges[i * 2 + 1];
	
	int* arrStart = &sorted[n1 * MAX_DEGREE];
	int retnVal = 1;
	while(retnVal != 0)
		retnVal = atomicCAS(arrStart, 0, n2);
}


//HACK: This will be incredibly slow  on CUDA!
__device__ int findNext(int* edges, int numEdge, int v, int* destination)
{
	int count = 0;

	for(int i = 0; i < numEdge * 2; i+=2)
	{
		if(edges[i] == v)
			destination[count++] = edges[i + 1];
	}

	return count;
}

__device__ void pushQueue(int element, int* queue, int queueSize, int* head, int* tail)
{
	queue[*tail] = element;
	*tail = (*tail + 1) % queueSize;
}

__device__ int popQueue(int* queue, int queueSize, int* head, int* tail)
{
	int retn = queue[*head];
	*head = (*head + 1) % queueSize;
	
	return retn;
}

__device__ void pushStack(int element, int* stack, int* head)
{
	stack[*head] = element;
	*head = *head + 1;
}

__device__ int popStack(int* stack, int* head)
{
	*head = *head - 1;
	int retn = stack[*head];
	
	return retn;
}

__device__ int ELEMENTS = DEFAULT_ELE;
__device__ int S_SIZE;
__device__ int P_SIZE;
__device__ int PATH_SIZE;
__device__ int D_SIZE;
__device__ int Q_SIZE;

__device__ void doAlg(int numVert, int* edges, int numEdges, float* BC, int* glob, float* globDep)
{
	S_SIZE = numVert;
	P_SIZE = numVert;
	D_SIZE = numVert;
	Q_SIZE = numVert;
	PATH_SIZE = numVert;
	ELEMENTS = numVert;

	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.y + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	int PTR_OFFSET = idx * (S_SIZE + (P_SIZE * MAX_DEGREE) + D_SIZE + Q_SIZE + PATH_SIZE);
	
	int* S = &glob[PTR_OFFSET];
	int S_head = 0;
	PTR_OFFSET += S_SIZE;
	
	int* P = &glob[PTR_OFFSET];
	//Blank the previous items
	for(int i = 0; i < P_SIZE; i++)
		for(int j = 0; j < MAX_DEGREE; j++)
		{
			P[i + P_SIZE * j] = -1;
		}
	PTR_OFFSET += P_SIZE * MAX_DEGREE;

	int* pathCount = &glob[PTR_OFFSET];
	for(int i = 0; i < PATH_SIZE; i++)
	{
		pathCount[i] = 0;
	}
	pathCount[idx] = 1;
	PTR_OFFSET += PATH_SIZE;

	int* d = &glob[PTR_OFFSET];
	for(int i = 0; i < D_SIZE; i++)
	{
		d[i] = -1;
	}
	d[idx] = 0;
	PTR_OFFSET += D_SIZE;
	
	int* Q = &glob[PTR_OFFSET];
	int Q_head = 0;
	int Q_tail = 0;
	PTR_OFFSET += Q_SIZE;
	
	pushQueue(idx, Q, Q_SIZE, &Q_head, &Q_tail);

	while(Q_head != Q_tail)
	{
		int v = popQueue(Q, Q_SIZE, &Q_head, &Q_tail);
		pushStack(v, S, &S_head);

		int w[MAX_DEGREE];
		int edgeCount = findNext(edges, numEdges, v, w);

		for(int i = 0; i < edgeCount; i++)
		{
			int wNode = w[i];
			if(d[wNode] < 0)
			{
				pushQueue(wNode, Q, Q_SIZE, &Q_head, &Q_tail);
				d[wNode] = d[v] + 1;
			}
			
			if(d[wNode] == d[v] + 1)
			{
				pathCount[wNode] = pathCount[wNode] + pathCount[v];
				
				//Append v to the PrevNode list
				for(int j = 0; j < MAX_DEGREE; j++)
				{
					if(P[wNode + P_SIZE * j] < 0)
					{
						P[wNode + P_SIZE * j] = v;
						break;
					}
				}
			}
		}

	}
	
	float* dep = &globDep[idx * ELEMENTS];
	
	while(S_head > 0)
	{
		int w = popStack(S, &S_head);
		
		//Loop through each v in P[w]
		for(int i = 0; i < MAX_DEGREE; i++)
		{
			int v = P[w + P_SIZE * i];
			if(v < 0) continue;

			dep[v] = dep[v] + ((float)pathCount[v]/(float)pathCount[w]) * (1 + dep[w]);
		}
		
		if(w != idx)
		{
			atomicAdd(&BC[w], dep[w]);
		}
	}
	
}

__global__ void betweennessCentrality(int numVert, int numEdges, int *edges, float* BC, int* glob, float* dep)
{
	extern __shared__ int path[];
	
	//sortEdges(edges, path);
	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.x + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	if(idx >= numVert) return;

	BC[idx] = 0.0f;

	__syncthreads();

	doAlg(numVert, edges, numEdges, BC, glob, dep);

		
}

int main(int argc, char* argv[])
{
	int elements = DEFAULT_ELE;

	//cudaProfilerStart();
	int *d_mem;
	int *h_edge;
	int *d_edge;
	float *d_bc;
	float *h_bc;
	int *d_glob;
	float *d_dep;

	int numVert = elements;
	int numEdge = elements - 1;
	

	if(argc < 2)
	{

		FILE *grFile;
		grFile = fopen("test.gr", "w");
		fprintf(grFile, "p %d %d d u 0\n", numVert, numEdge);
		h_edge = (int*)malloc(sizeof(int) * numEdge * 2);
		
		//Init edges
		for(int i = 0; i < numEdge; i++)
		{
			h_edge[i * 2] = i % numVert;
			h_edge[i * 2 + 1] = (i + 1) % numVert;
			fprintf(grFile, "%d %d\n", h_edge[i * 2], h_edge[i * 2 + 1]);
		}
		fclose(grFile);
	}
	else
	{
		FILE *grFile;
		char buff[1024];
		int edgeCnt = 0;
		grFile = fopen(argv[1], "r");
		while(!feof(grFile))
		{
			fgets(buff, 1024, grFile);
			if(feof(grFile))break;

			//This is the  "problem" line
			if(buff[0] == 'p')
			{
				char* token = strtok(buff, " ");

				token = strtok(NULL, " ");
				token = strtok(NULL, " ");
				numVert = atoi(token);
				
				token = strtok(NULL, " ");
				numEdge = atoi(token) * 2;

				h_edge = (int*)malloc(sizeof(int) * numEdge * 2);

			}
			else if(buff[0] == '#' || buff[0] == 'c')
				continue;
			else if(buff[0] == 'a')
			{
				char* token = strtok(buff, " ");

				//Skip 'a'
				token = strtok(NULL, " ");
				int e1 = atoi(token) - 1;
				h_edge[edgeCnt] = e1;
				h_edge[edgeCnt + 3] = e1;
				token = strtok(NULL, " ");
				int e2 = atoi(token) - 1;
				h_edge[edgeCnt + 1] = e2;
				h_edge[edgeCnt + 2] = e2;
				edgeCnt += 4;
			}
		}
	}
	h_bc = (float*)malloc(sizeof(float) * numVert);
	hipMalloc((void**)&d_mem, sizeof(int) * numVert);
	
	hipMalloc((void**)&d_edge, sizeof(int) * numEdge * 2);

	hipMalloc((void**)&d_bc, sizeof(float) * numVert);

	hipMalloc((void**)&d_glob, sizeof(int) * numVert * numVert * 8);
	hipMalloc((void**)&d_dep, sizeof(float) * numVert * numVert);
	hipMemcpy(d_edge, h_edge, sizeof(int) * numEdge * 2, hipMemcpyHostToDevice);
	
	dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	int gridSize = ceil(numVert / (float)(BLOCK_WIDTH * BLOCK_HEIGHT));
	dim3 grid(gridSize);
	//test<<<grid,block>>>(d_mem);
	betweennessCentrality<<<grid,block>>>(numVert, numEdge, d_edge, d_bc, d_glob, d_dep);
	hipError_t error = hipGetLastError();
	
	int* h_mem = (int*)malloc(sizeof(int) * numVert);
	hipMemcpy(h_mem, d_mem, sizeof(int) * numVert, hipMemcpyDeviceToHost);
	hipMemcpy(h_bc, d_bc, sizeof(float) * numVert, hipMemcpyDeviceToHost);
	

	for(int i = 0; i < numVert; i++)
	{
		cout << h_bc[i] << endl;
	}
	//cout<<elements<<endl;
	
	//cudaProfilerStop();
	
	hipDeviceReset();
	cout << hipGetErrorString(error) << endl;
	
	return 0;
}
